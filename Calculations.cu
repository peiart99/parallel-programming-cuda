#include "hip/hip_runtime.h"
#include "Calculations.h"

void calculateAnswer(int array_length, int reach, float *data_array, float *out_array)
{
    for(int i {0}; i < (array_length - (2 * reach)); i++)
    {
        for(int j {0}; j < (array_length - (2 * reach)); j++)
        {
            calculateSquare(array_length, reach, i, j, data_array, out_array);
        }
    }
}


void calculateSquare(int array_length, int reach, int central_i, int central_j, float *data_array, float *out_array)
{
    for(int i {central_i}; i < 2 * reach + 1 + central_i; i++)
    {
        for(int j {central_j}; j < 2 * reach + 1 + central_j; j++)
        {
            out_array[(central_i * (array_length - (2 * reach))) + central_j] += data_array[(i * array_length) + j];
        }
    }
}

__global__ void deviceCalculateAnswer(int array_length, int reach, int elements_per_thread, float *data_array, float *out_array)
{
    int tid = threadIdx.x * elements_per_thread;
    int bid = threadIdx.y;
    for(int k {0}; k < elements_per_thread; k++)
    {
        for(int i {bid}; i < 2 * reach + 1 + bid; i++)
        {
            for(int j {tid}; j < 2 * reach + 1 + tid; j++)
            {
                out_array[(bid * (array_length - (2 * reach))) + tid] += data_array[(i * array_length) + j];
            }
        }
        tid++;
    }

}

__global__ void deviceCalculateAnswer_test(int array_length, int reach, int elements_per_thread, float *data_array, float *out_array)
{
    int tid = threadIdx.x * elements_per_thread;
    int bid = threadIdx.y;
    for(int k {0}; k < elements_per_thread; k++)
    {
        for(int j {tid}; j < 2 * reach + 1 + tid; j++)
        {
            for(int i {bid}; i < 2 * reach + 1 + bid; i++)
            {
                out_array[(bid * (array_length - (2 * reach))) + tid] += data_array[(i * array_length) + j];
            }
        }
        tid++;
    }

}

__global__ void deviceCalculateAnswer_shared(int array_length, int reach, int elements_per_thread, int data_move_per_thread, float *data_array, float *out_array)
{
    int tid = threadIdx.x * elements_per_thread;
    int bid = threadIdx.y;
    //printf("block: %d %d %d %d %d\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, blockDim.z);
    extern __shared__ float shared_data[];

    int shared_index = (threadIdx.y * (array_length - (2 * reach)) * data_move_per_thread) + (threadIdx.x * data_move_per_thread);

    //printf("[%d][%d] : %d\n", threadIdx.y, threadIdx.x, shared_index);

    for(int i {shared_index}; i < (shared_index + data_move_per_thread); i++)
    {
        if(i < (array_length * array_length))
        {
            shared_data[i] = data_array[i];
            //printf("elements: %d\n", elements_per_thread);
            //printf("index: %d data: %f shared: %f\n", i, data_array[i], shared_data[i]);
        }else
        {
            //break;
        }
    }

    __syncthreads();

    for(int k {0}; k < elements_per_thread; k++)
    {
        for(int i {bid}; i < 2 * reach + 1 + bid; i++)
        {
            for(int j {tid}; j < 2 * reach + 1 + tid; j++)
            {
                out_array[(bid * (array_length - (2 * reach))) + tid] += shared_data[(i * array_length) + j];
            }
        }
        tid++;
    }

}