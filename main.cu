#include <iostream>
#include "Generator.h"
#include "Array.h"
#include "Calculations.h"
#include <chrono>

void calculate(int array_length, int reach, float *data_array, float *out_array, float *device_data_array, float *device_out_array, size_t bytes_data, size_t bytes_out)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << prop.name << std::endl;
    std::cout << prop.major << "." << prop.minor << std::endl;
    std::cout << prop.multiProcessorCount << std::endl;
    std::cout << prop.maxThreadsPerBlock << std::endl;
    std::cout << prop.maxBlocksPerMultiProcessor << std::endl;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //printArray(array_length, data_array);
    auto start_cpu = std::chrono::steady_clock::now();
    calculateAnswer(array_length, reach, data_array, out_array);
    auto stop_cpu = std::chrono::steady_clock::now();
    std::cout << "ANSWER CALCULATED IN " << std::chrono::duration_cast<std::chrono::microseconds>(stop_cpu - start_cpu).count() << "us" << std::endl;
    //printArray(array_length - (2 * reach), out_array);

    // define the dimensions of the grid and thread blocks
    dim3 threads_per_block(array_length - (2 * reach),array_length - (2 * reach));
    dim3 number_of_blocks((array_length - (2 * reach)) / threads_per_block.x, (array_length - (2 * reach)) / threads_per_block.y);

    // cuda kernel call
    hipEventRecord(start);
    deviceCalculateAnswer<<<number_of_blocks, threads_per_block>>>(array_length, reach, device_data_array, device_out_array);
    hipEventRecord(stop);

    // wait for the device to finish executing before continuing
    hipDeviceSynchronize();

    std::cout << "Error: " << hipGetErrorString(hipGetLastError()) << '\n';
    // copy the result stored in the array in the device to a host array
    hipMemcpy(out_array, device_out_array, bytes_out, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU ANSWER CALCULATED IN: " << milliseconds * 1000.0f << " us" << std::endl;
    //printArray(array_length - (2 * reach), out_array);
}

int main()
{
    int array_length {10};
    int reach {2};
    auto *data_array = new float[array_length * array_length];
    auto *out_array = new float[(array_length - (2 * reach)) * (array_length - (2 * reach))];
    float *device_data_array, *device_out_array;

    // calculate the size in bytes of host arrays
    size_t bytes_data {(array_length * array_length) * sizeof(float)};
    size_t bytes_out {((array_length - (2 * reach)) * (array_length - (2 * reach))) * sizeof(float)};

    // allocate memory on the device equal to host data that's going to be used by the device
    hipMalloc(&device_data_array, bytes_data);
    hipMalloc(&device_out_array, bytes_out);

    // fill the data array (setting the last parameter named "debug" to "true" fills the array with 1's to make debugging easy)
    populateArray(array_length, data_array, false);

    // copy the host data array's content into the memory previously allocated on the device
    hipMemcpy(device_data_array, data_array, bytes_data, hipMemcpyHostToDevice);

    calculate(array_length, reach, data_array, out_array, device_data_array, device_out_array, bytes_data, bytes_out);

    // free the heap allocated to dynamically created host arrays
    delete[] data_array;
    delete[] out_array;

    // free the memory allocated to arrays on the device
    hipFree(device_data_array);
    hipFree(device_out_array);
    return 0;
}