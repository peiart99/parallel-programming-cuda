
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <random>
#include <chrono>

void calculateSquare(int array_length, int reach, int central_i, int central_j, float *data_array, float *out_array);

double generateRandomFloat(float min_value, float max_value)
{
    std::default_random_engine eng;
    unsigned long int t = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    eng.seed(t);
    static std::mt19937 gen(eng());
    std::uniform_real_distribution<> dist(min_value,max_value);
    return dist(gen);
}

void printArray(int array_length, float *array)
{
    for(int i {0}; i < array_length; i++)
    {

        for(int j {0}; j < array_length; j++)
        {
            std::cout << array[(array_length * i) + j] << ", ";
        }
        std::cout << std::endl;
    }
}

void populateArray(int array_length, float *array, bool zero)
{
    for(int i {0}; i < array_length; i++)
    {
        for(int j {0}; j < array_length; j++)
        {
            if(!zero)
            {
                //array[i][j] = generateRandomFloat(1.0f, 100.0f);
                array[(array_length * i) + j] = 1;
            }else
            {
                array[(array_length * i) + j] = 0;
            }

        }
    }
}

void calculateAnswer(int array_length, int reach, float *data_array, float *out_array)
{
    for(int i {0}; i < (array_length - (2 * reach)); i++)
    {
        for(int j {0}; j < (array_length - (2 * reach)); j++)
        {
            calculateSquare(array_length, reach, i, j, data_array, out_array);
        }
    }
}

void calculateSquare(int array_length, int reach, int central_i, int central_j, float *data_array, float *out_array)
{
    int upper_left_index_j {0};
    int upper_left_index_i {0};
    int square_length = 2 * reach + 1;
    int lower_edge {0};
    int right_edge {0};
    int diff_j {0};
    int diff_i {0};
    if(central_j - reach >= 0)
    {
        upper_left_index_j = central_j - reach;
    }else
    {
        upper_left_index_j = 0;
    }
    diff_j = upper_left_index_j - (central_j - reach);
    right_edge = upper_left_index_j - diff_j + square_length;
    if(right_edge > array_length)
    {
        right_edge = array_length;
    }

    if(central_i - reach >= 0)
    {
        upper_left_index_i = central_i - reach;
    }else
    {
        upper_left_index_i - 0;
    }
    diff_i = upper_left_index_i - (central_i - reach);
    lower_edge = upper_left_index_i - diff_i + square_length;

    for(int i {upper_left_index_i}; i < lower_edge; i++)
    {
        for(int j {upper_left_index_j}; j < right_edge; j++)
        {
            out_array[(central_i * (array_length - (2 * reach))) + central_j] += data_array[(i * array_length) + j];
        }
    }
}

__global__ void deviceCalculateAnswer(int array_length, int reach, float *data_array, float *out_array)
{
    //printf("block id: %d thread id: %d\n", blockIdx.x, threadIdx.x);
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int upper_left_index_j {0};
    int upper_left_index_i {0};
    int square_length = 2 * reach + 1;
    int lower_edge {0};
    int right_edge {0};
    int diff_j {0};
    int diff_i {0};
    if(tid - reach >= 0)
    {
        upper_left_index_j = tid - reach;
    }else
    {
        upper_left_index_j = 0;
    }
    diff_j = upper_left_index_j - (tid - reach);
    right_edge = upper_left_index_j - diff_j + square_length;
    if(right_edge > array_length)
    {
        right_edge = array_length;
    }

    if(bid - reach >= 0)
    {
        upper_left_index_i = bid - reach;
    }else
    {
        upper_left_index_i - 0;
    }
    diff_i = upper_left_index_i - (bid - reach);
    lower_edge = upper_left_index_i - diff_i + square_length;

    for(int i {upper_left_index_i}; i < lower_edge; i++)
    {
        for(int j {upper_left_index_j}; j < right_edge; j++)
        {
            out_array[(bid * (array_length - (2 * reach))) + tid] += data_array[(i * array_length) + j];
        }
    }
}

int main()
{
    int array_length {10};
    int reach {2};

    auto *data_array = new float[array_length * array_length];
    auto *out_array = new float[(array_length - (2 * reach)) * (array_length - (2 * reach))];
    float *device_data_array, *device_out_array;
    size_t bytes_data {(array_length * array_length) * sizeof(float)};
    size_t bytes_out {((array_length - (2 * reach)) * (array_length - (2 * reach))) * sizeof(float)};
    hipMalloc(&device_data_array, bytes_data);
    hipMalloc(&device_out_array, bytes_out);
    populateArray(array_length, data_array, false);
    populateArray(array_length - (2 * reach), out_array, true);
    hipMemcpy(device_data_array, data_array, bytes_data, hipMemcpyHostToDevice);
    printArray(array_length, data_array);
    calculateAnswer(array_length, reach, data_array, out_array);
    std::cout << "ANSWER" << std::endl;
    printArray(array_length - (2 * reach), out_array);

    deviceCalculateAnswer<<<array_length - (2 * reach), array_length - (2 * reach)>>>(array_length, reach, device_data_array, device_out_array);
    hipDeviceSynchronize();
    hipMemcpy(out_array, device_out_array, bytes_out, hipMemcpyDeviceToHost);
    std::cout << "GPU ANSWER" << std::endl;
    printArray(array_length - (2 * reach), out_array);


    delete[] data_array;
    delete[] out_array;
    hipFree(device_data_array);
    hipFree(device_out_array);
    return 0;
}